#include "hip/hip_runtime.h"
#include <netdb.h> 
#include <netinet/in.h> 
#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <sys/socket.h> 
#include <sys/types.h>
#include <unistd.h>
#include <time.h> 
#define PORT 8080 
#define SA struct sockaddr 

// nvcc cuda.cu -o Cuda, ./Cuda

int create() {
    int sockfd, connfd; 
    socklen_t len;
    struct sockaddr_in servaddr, cli; 
  
    // socket create and verification 
    sockfd = socket(AF_INET, SOCK_STREAM, 0); 
    if (sockfd == -1) { 
        printf("socket creation failed...\n"); 
        exit(0); 
    } 
    else
        printf("Socket successfully created..\n"); 
    
    //if (setsockopt(sockfd, SOL_SOCKET, SO_REUSEADDR, &(int){1}, sizeof(int)) < 0)
	printf("setsockport(SO_REUSEADDR) failed \n");

    bzero(&servaddr, sizeof(servaddr)); 
  
    // assign IP, PORT 
    servaddr.sin_family = AF_INET; 
    servaddr.sin_addr.s_addr = htonl(INADDR_ANY); 
    servaddr.sin_port = htons(PORT); 
  
    // Binding newly created socket to given IP and verification 
    if ((bind(sockfd, (SA*)&servaddr, sizeof(servaddr))) != 0) { 
        printf("socket bind failed...\n"); 
        exit(0); 
    } 
    else
        printf("Socket successfully binded..\n"); 
  
    // Now server is ready to listen and verification 
    if ((listen(sockfd, 5)) != 0) { 
        printf("Listen failed...\n"); 
        exit(0); 
    } 
    else
        printf("Server listening..\n"); 
    len = sizeof(cli); 
  
    // Accept the data packet from client and verification 
    connfd = accept(sockfd, (SA*)&cli, &len); 
    if (connfd < 0) { 
        printf("server acccept failed...\n"); 
        exit(0); 
    } 
    else
        printf("server acccept the client...\n"); 
    
    return connfd;
}

__global__ void process(int n, int height, int width, unsigned char *in, unsigned char *out) {
    int k;
    for(k = 0; k < width; k++) {
        int current = blockIdx.x * width * height + threadIdx.x * width + k;
        if (k + threadIdx.x < width) {
            out[current] = in[current + threadIdx.x];
        } else {
            out[current] = in[current + threadIdx.x - width];
        }
    }
}
  
// Driver function 
int main(int argc, char **argv) 
{ 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int sockfd;
    int n, width, height;  

    unsigned char *in = NULL;
    unsigned char *cuda_in = NULL;
    unsigned char *cuda_out = NULL;
    unsigned char *out = NULL;
    //struct timeval start, end;

    sockfd = create();
    
    read(sockfd, &n, sizeof(n));
    read(sockfd, &width, sizeof(width));
    read(sockfd, &height, sizeof(height));
    n = ntohl(n);
    width = ntohl(width);
    height = ntohl(height);
        
    printf("Received header:\n Number of matrices = %d\n Width of each = %d\n Height of each = %d\n", n, width, height);

    in = (unsigned char*) malloc(sizeof(unsigned char) * n * width * height);
    out = (unsigned char*) malloc(sizeof(unsigned char) * n * width * height);

    for (size_t i = 0; i < n; i++) {
        read(sockfd, in + i * height * width, sizeof(unsigned char) * height * width);
    }

    hipMalloc(&cuda_in, sizeof(unsigned char) * n * width * height);
    hipMalloc(&cuda_out, sizeof(unsigned char) * n * width * height);
    hipMemcpy(cuda_in, in, sizeof(unsigned char) * n * width * height, hipMemcpyHostToDevice);
    
    hipEventRecord(start);
    process << < n, height >> > (n, height, width, cuda_in, cuda_out);
    hipEventRecord(stop);
    

    hipMemcpy(out, cuda_out, sizeof(unsigned char) * n * width * height, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);

    hipFree(cuda_in);
    hipFree(cuda_out);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\n\nTIME %f\n\n", milliseconds);s

    for (size_t i = 0; i < n; i++) {
        printf(".");
        write(sockfd, out + i * height * width, sizeof(unsigned char) * height * width);
    }
        
    write(sockfd, &milliseconds, sizeof(milliseconds));
    printf("Result send.\n");
    free(in);
    free(out);
    close(sockfd); 
    
    return 0;
} 
