#include "hip/hip_runtime.h"
#include <netdb.h> 
#include <netinet/in.h> 
#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <sys/socket.h> 
#include <sys/types.h>
#include <unistd.h>
#include <time.h> 
#define PORT 8080 
#define SA struct sockaddr 

// nvcc cuda.cu -o Cuda, ./Cuda

int create() {
    int sockfd, connfd; 
    socklen_t len;
    struct sockaddr_in servaddr, cli; 
  
    // socket create and verification 
    sockfd = socket(AF_INET, SOCK_STREAM, 0); 
    if (sockfd == -1) { 
        printf("socket creation failed...\n"); 
        exit(0); 
    } 
    else
        printf("Socket successfully created..\n"); 
    
    //if (setsockopt(sockfd, SOL_SOCKET, SO_REUSEADDR, &(int){1}, sizeof(int)) < 0)
	printf("setsockport(SO_REUSEADDR) failed \n");

    bzero(&servaddr, sizeof(servaddr)); 
  
    // assign IP, PORT 
    servaddr.sin_family = AF_INET; 
    servaddr.sin_addr.s_addr = htonl(INADDR_ANY); 
    servaddr.sin_port = htons(PORT); 
  
    // Binding newly created socket to given IP and verification 
    if ((bind(sockfd, (SA*)&servaddr, sizeof(servaddr))) != 0) { 
        printf("socket bind failed...\n"); 
        exit(0); 
    } 
    else
        printf("Socket successfully binded..\n"); 
  
    // Now server is ready to listen and verification 
    if ((listen(sockfd, 5)) != 0) { 
        printf("Listen failed...\n"); 
        exit(0); 
    } 
    else
        printf("Server listening..\n"); 
    len = sizeof(cli); 
  
    // Accept the data packet from client and verification 
    connfd = accept(sockfd, (SA*)&cli, &len); 
    if (connfd < 0) { 
        printf("server acccept failed...\n"); 
        exit(0); 
    } 
    else
        printf("server acccept the client...\n"); 
    
    return connfd;
}

__global__ void process(long grid_size, long treads, int n, int height, int width, unsigned char *in, unsigned char *out) {
    int i, j, k;
    for(i = 0; i < n; i++) {     
        for(j = 0; j < height; j++) {
            for(k = 0; k < width; k++) {
                int current = i * width * height + j * width + k;
                if (k + j < width) {
                    out[current] = in[current + j];
                } else {
                    out[current] = in[current + j - width];
                }
            }
        }
    }
}
  
// Driver function 
int main(int argc, char **argv) 
{ 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int sockfd;
    int n, width, height;
    
    long grid_size = 2;
    long treads_size = 2;    

    unsigned char *in = NULL;
    unsigned char *cuda_in = NULL;
    unsigned char *cuda_out = NULL;
    unsigned char *out = NULL;
    //struct timeval start, end;

    sockfd = create();
    
    read(sockfd, &n, sizeof(n));
    read(sockfd, &width, sizeof(width));
    read(sockfd, &height, sizeof(height));
    n = ntohl(n);
    width = ntohl(width);
    height = ntohl(height);
        
    printf("Received header:\n Number of matrices = %d\n Width of each = %d\n Height of each = %d\n", n, width, height);

    in = (unsigned char*) malloc(sizeof(unsigned char) * n * width * height);
    out = (unsigned char*) malloc(sizeof(unsigned char) * n * width * height);

    for (size_t i = 0; i < n; i++) {
        read(sockfd, in + i * height * width, sizeof(unsigned char) * height * width);
    }
    
    //gettimeofday(&start, NULL);

    hipMalloc(&cuda_in, sizeof(unsigned char) * n * width * height);
    hipMalloc(&cuda_out, sizeof(unsigned char) * n * width * height);

    hipMemcpy(cuda_in, in, sizeof(unsigned char) * n * width * height, hipMemcpyHostToDevice);
    hipEventRecord(start);

    process << < grid_size, treads_size >> > (grid_size, treads_size, n, height, width, cuda_in, cuda_out);

    hipEventRecord(stop);

    hipMemcpy(out, cuda_out, sizeof(unsigned char) * n * width * height, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    hipFree(cuda_in);
    hipFree(cuda_out);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\n\nTIME %f\n\n", milliseconds);

        /*gettimeofday(&end, NULL);
        printf("Sending result...\n");
        double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + 
         end.tv_usec - start.tv_usec) / 1.e6;
	printf("\nElapsed: %lf ms\n", delta);*/

    for (size_t i = 0; i < n; i++) {
        write(sockfd, out + i * height * width, sizeof(unsigned char) * height * width);
    }
        
    write(sockfd, &milliseconds, sizeof(milliseconds));
    printf("Result send.\n");
    free(in);
    free(out);
    close(sockfd); 
    
    return 0;
} 
